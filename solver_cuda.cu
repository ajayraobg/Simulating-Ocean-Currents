#include "hip/hip_runtime.h"
#include "grid.h"
#include "solver_cuda.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

static void fail(char const *message);
__global__ void red_black_ordering(float *d_grid, int grid_row, int value, float *d_diff1);

void Solver_cuda::simulate_eqn_solver(){
	int done = 0;
	float diff = 0.0;
	struct timeval start_time, end_time; // Variables to capture timing details
	int size = ( row * col * sizeof(float) );
	hipError_t err = hipSuccess;

	// The code below is optional.  We are going to create a diff array instead of
	// accumulating the diffs in a variable.  If we didn't do that, we would need to
	// synchronize threads as they summed the diffs, and that's harder for beginners.
	float *diff1 = NULL;
	diff1 = (float*) malloc (size);
	for (int i= 0; i < row; i++) {
		for (int j = 0; j < col; j++) {
			diff1[i * col + j] = 0.0;
		}
	}

	int threads_per_block = 256;
	int blocks_per_grid = (int)( ( row*col + threads_per_block - 1) / threads_per_block);		
	float *d_grid = NULL;
	err = hipMalloc((void **)&d_grid, size);
	if(err != hipSuccess) cout<<"Failed to allocate grid memory"<<endl;
	err = hipMemcpy(d_grid,array,size,hipMemcpyHostToDevice);
	if(err != hipSuccess) cout<<"Failed to copy grid to device"<<endl;
	float *d_diff1 = NULL;
	err = hipMalloc((void **)&d_diff1, size);
	if(err != hipSuccess) cout<<"Failed to allocate diff memory"<<endl;
	err = hipMemcpy(d_diff1,diff1,size,hipMemcpyHostToDevice);
	if(err != hipSuccess) cout<<"Failed to copy diff to device"<<endl;

	// Calculate start time for equation solver
	gettimeofday(&start_time, NULL);
	
	while (!done){
		diff = 0.0;
		red_black_ordering <<< blocks_per_grid, threads_per_block >>> (d_grid, row, RED, d_diff1);
		hipDeviceSynchronize();		
		red_black_ordering <<< blocks_per_grid, threads_per_block >>> (d_grid, row, BLACK, d_diff1);
		hipDeviceSynchronize();
		err = hipMemcpy(diff1,d_diff1,size,hipMemcpyDeviceToHost);
		if(err != hipSuccess) cout<<"Failed to copy diff from device"<<endl;
		for (int i= 0; i < row; i++) 
		{
			for (int j = 0; j < col; j++) 
			{
				diff = diff + diff1[i * col + j];
			}
		}
		diff = diff/float (row*col);
		if(diff<tolerance) {done = 1;}
		else {done = 0;}		
	}

	gettimeofday(&end_time, NULL);
	// Print the final Timing Statistics
	print_timing_statistics(start_time,end_time);
	
	err = hipMemcpy(array,d_grid,size,hipMemcpyDeviceToHost);
	if(err != hipSuccess) cout<<"Failed to copy grid from device"<<endl;
	err = hipFree(d_diff1);
	if(err != hipSuccess) cout<<"Failed to free diff memory"<<endl;
	err = hipFree(d_grid);
	if(err != hipSuccess) cout<<"Failed to free grid memory"<<endl;
	free(diff1);
	hipDeviceReset();
}
//
//
__global__ void red_black_ordering(float *d_grid, int grid_row, int value, float *d_diff1)

{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	float temp = 0.0;
	int i;
	int row = index/grid_row;
	int col = index - (row * grid_row);
	int flag = 0;


	if(index<grid_row*grid_row)
	{
		if(!((row==0)|| (row==grid_row-1) || (col==0) || (col==grid_row-1))) 
		{
			if(!value)
			{
				if(((row%2==1) && (col%2==1)) || ((row%2==0) && (col%2==0)))
				{
					temp = d_grid[row*grid_row + col];
					d_grid[row*grid_row + col] = 0.2 * (d_grid[row*grid_row + col] + d_grid[row*grid_row + col +1] + d_grid[row*grid_row + col - 1] + d_grid[(row-1)*grid_row + col] + d_grid[(row+1)*grid_row + col]);
					d_diff1[index] = fabs(temp - d_grid[row*grid_row + col]);
				}				
			}
			else
			{
				if(((row%2==1) && (col%2==0)) || ((row%2==0) && (col%2==1)))
				{
					temp = d_grid[row*grid_row + col];
					d_grid[row*grid_row + col] = 0.2 * (d_grid[row*grid_row + col] + d_grid[row*grid_row + col +1] + d_grid[row*grid_row + col - 1] + d_grid[(row-1)*grid_row + col] + d_grid[(row+1)*grid_row + col]);
					d_diff1[index] = fabs(temp - d_grid[row*grid_row + col]);
				}
			}
		}
	}
	

}

